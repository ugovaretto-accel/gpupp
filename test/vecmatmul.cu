
#include <hip/hip_runtime.h>
#ifdef DOUBLE__
typedef double real_t;
#else
typedef float real_t;
#endif

//#define COLUMN //2x speed increase!

typedef unsigned uint;
extern "C" __global__ void VecMatMul( const real_t* M,
									  uint width,
									  uint height,
									  const real_t* V,
									  real_t* W )
{
 
#ifdef COLUMN // vector * matrix
  uint c = blockIdx.x * blockDim.x + threadIdx.x;
  //if( c >= height ) return;
  const real_t* column = M + c;
  real_t dp = 0.f;
  for( uint r = 0; r < height * width; r += width )
  {
    dp += column[ r ] * V[ c ];
  }
  W[ c ] = dp;
#else // matrix * vector
  uint r = blockIdx.x * blockDim.x + threadIdx.x;
  //if( r >= width ) return;
  const real_t* row = M + r * width;
  real_t dp = 0.f;
  for( uint c = 0; c != width; ++c )
  {
    dp += row[ c ] * V[ c ];
  }
  W[ r ] = dp;
 #endif 
}                         
                         
                                      
  